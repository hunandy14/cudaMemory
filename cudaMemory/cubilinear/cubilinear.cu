#include "hip/hip_runtime.h"
﻿/***************************************************************************************
Name :
Date : 2018/01/08
By   : CharlotteHonG
Final: 2018/01/08
***************************************************************************************/
#include <utility>
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
#include <algorithm>
#include <cmath>
using namespace std;

#include "cubilinear.hpp"
#define BLOCK_DIM 16.0
#define BLOCK_DIM_X 32.0
#define BLOCK_DIM_Y 8.0

using uch = unsigned char;

//======================================================================================
// 快速線性插值_核心
__device__ __host__ static inline
void fast_Bilinear(const uch* src, int w, int h,
	uch* p, double y, double x)
{
	int srcW = w;
	int srcH = h;

	// 起點
	int _x = (int)x;
	int _y = (int)y;
	// 左邊比值
	double l_x = x - (double)_x;
	double r_x = 1.f - l_x;
	double t_y = y - (double)_y;
	double b_y = 1.f - t_y;

	// 計算RGB
	double R , G, B;
	int x2 = (_x+1) > srcW -1? srcW -1: _x+1;
	int y2 = (_y+1) > srcH-1? srcH-1: _y+1;

	R  = (double)src[(_y * srcW + _x) *3 + 0] * (r_x * b_y);
	G  = (double)src[(_y * srcW + _x) *3 + 1] * (r_x * b_y);
	B  = (double)src[(_y * srcW + _x) *3 + 2] * (r_x * b_y);
	R += (double)src[(_y * srcW + x2) *3 + 0] * (l_x * b_y);
	G += (double)src[(_y * srcW + x2) *3 + 1] * (l_x * b_y);
	B += (double)src[(_y * srcW + x2) *3 + 2] * (l_x * b_y);
	R += (double)src[(y2 * srcW + _x) *3 + 0] * (r_x * t_y);
	G += (double)src[(y2 * srcW + _x) *3 + 1] * (r_x * t_y);
	B += (double)src[(y2 * srcW + _x) *3 + 2] * (r_x * t_y);
	R += (double)src[(y2 * srcW + x2) *3 + 0] * (l_x * t_y);
	G += (double)src[(y2 * srcW + x2) *3 + 1] * (l_x * t_y);
	B += (double)src[(y2 * srcW + x2) *3 + 2] * (l_x * t_y);

	p[0] = (unsigned char) R;
	p[1] = (unsigned char) G;
	p[2] = (unsigned char) B;
}
// 快速線性插值
__global__ 
void cuWarpScale_kernel(const uch* src, uch* dst, 
	int w, int h, double ratio)
{
	int srcW=w;
	int srcH=h;

	int dstW = (int)((srcW * ratio) +0.5);
	int dstH = (int)((srcH * ratio) +0.5);

	// 縮小的倍率
	double r1W = ((double)srcW )/(dstW);
	double r1H = ((double)srcH)/(dstH);
	// 放大的倍率
	double r2W = (srcW -1.0)/(dstW -1.0);
	double r2H = (srcH-1.0)/(dstH-1.0);
	// 縮小時候的誤差
	double deviW = ((srcW-1.0)  - (dstW -1.0)*(r1W)) /dstW;
	double deviH = ((srcH-1.0) - (dstH-1.0)*(r1H)) /dstH;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(j < dstH && i < dstW) { // 會多跑一點點要擋掉
			double srcY, srcX;
			if (ratio < 1.0) {
				srcX = i*(r1W+deviW);
				srcY = j*(r1H+deviH);
			} else if (ratio >= 1.0) {
				srcX = i*r2W;
				srcY = j*r2H;
			}
			// 獲取插補值
			unsigned char* p = &dst[(j*dstW+ i) *3];
			fast_Bilinear(src, w, h, p, srcY, srcX);
		
	}
}
// GPU 線性插值
__host__
void WarpScale_rgb(const cuImgData & uSrc, cuImgData & uDst, double ratio) {
	// 設置大小
	int dstW = (int)((uSrc.width  * ratio) +0.5);
	int dstH = (int)((uSrc.height * ratio) +0.5);
	// 不相同則resize
	if(uDst.width != dstW || uDst.height != dstH || uDst.bits != uSrc.bits) 
		uDst.resize(dstW, dstH, uSrc.bits);
	// 設置執行緒
	dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 grid(ceil(dstW / BLOCK_DIM_X), ceil(dstH / BLOCK_DIM_Y));
	// 執行 kernel
	cuWarpScale_kernel <<< grid, block >>> (uSrc, uDst, uSrc.width, uSrc.height, ratio);
}
// 測試 cuWarpScale_kernel
__host__
void cuWarpScale_kernel_test(const basic_ImgData & src, basic_ImgData & dst, double ratio){
	Timer t;
	// 初始化空間
	//t.start();
	// 初始化 dst
	dst.width  = (int)((src.width  * ratio) +0.5);
	dst.height = (int)((src.height * ratio) +0.5);
	dst.bits   = src.bits;
	dst.raw_img.resize(dst.width * dst.height * dst.bits>>3);
	//t.print("  resize");

	// 要求GPU空間
	//t.start();
	cuMem<uch> gpuSrc(src.raw_img.size());
	//t.print("  cudamalloc gpuSrc");
	//t.start();
	cuMem<uch> gpuDst(dst.raw_img.size());
	//t.print("  cudamalloc gpuDst");

	// 複製資料
	// t.start();/
	gpuSrc.memcpyIn(src.raw_img.data(), src.raw_img.size());
	// t.print("  memcpyIn");

	// 設置執行緒
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	dim3 grid(ceil(dst.width / BLOCK_DIM), ceil(dst.width / BLOCK_DIM));

	// 執行 kernel
	// t.start();
	cuWarpScale_kernel <<< grid, block >>> (gpuSrc, gpuDst, src.width, src.height, ratio);
	// t.print("  kernel");

	// 複製資料
	// t.start();
	gpuDst.memcpyOut(dst.raw_img.data(), dst.raw_img.size());
	// t.print("  memcpyOut");


	// t.start();
	gpuDst.~cuMem<uch>();
	// t.print("  dctor");
}
